#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
//#include "MDR.h"
//#include "MDR_kernel.cu"
//#include "MDR.cu"

#define IDX(i,j,ld) (((i)*(ld))+(j))
#define imin(a,b) (a<b?a:b)

//const int GSx = imin( 32, (NUMCOMBS+BSx-1) / BSx );

int NSNPS;
int NIND;
float THR = -1.0;
int BSx = 256;
int NUMCOMBS;
//int GSx = ((NUMCOMBS+BSx-1) / BSx );
int GSx = 12500;
int ORDER = 3;
int CV = -1;
char* phenoFile;
char* genoFile;
char* outputFile;
char* combFile;

#define mat_SNP_size (NIND * NSNPS * sizeof(int))
#define v_pheno_size (NIND * sizeof(int))
#define output_size (NUMCOMBS * CV * 2 * sizeof(float)) //2 is one for train and one for test
#define combinations_size (NUMCOMBS * ORDER * sizeof(int))
#define indices_size (NIND * sizeof(int))

#define TESTCOMB 999


struct controlscases {
int controls;
int cases;
};

/*
__device__ int dev_rand() {
	return rand();
}
*/

//#include "MDR.h"
//#include "MDR.cu"

__constant__ int dev_v_pheno[6000];
__constant__ int dev_cv_indices[6000];
	

__global__ void MDR( int* dev_SNP_values, float* dev_output, int* dev_combinations, int NSNPS, int NIND, float THR, int NUMCOMBS, int ORDER, int CV ) {
    
    	
	//printf(" %d + %d * %d :", threadIdx.x, blockIdx.x, blockDim.x);
	//__shared__ float cache[BS][threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//printf(" %d ", tid);
	//int* thread_combination = (int*)malloc(ORDER * sizeof(int));
	int thread_combination[3]; //a combination (thread level)
	//retrieve the combination indices
	for (int i=0; i< ORDER; i++) {
		*(&thread_combination[0] + i) = *(dev_combinations + tid * ORDER + i);
	}
	//printf("thread with tid %d is assigned combination: <%d, %d, %d>\n", tid, thread_combination[0], 
	//thread_combination[1], thread_combination[2]); 
	
	//retrieve the genotype of each snp in the combination, from SNPvalues, for ALL individuals
	//int* thread_geno = (int*)malloc(NIND * ORDER * sizeof(int));
	int thread_geno[18000];
	for (int i=0; i< ORDER; i++) {
		for (int j=0; j< NIND; j++) {
			*(&thread_geno[0] + i * NIND + j) = *(dev_SNP_values + NIND * *(&thread_combination[0] + i) + j);
		}
	}
	
	if (tid == TESTCOMB){
			for (int i=0; i< 20; i++) {
				printf("thread %d %d-th-snp's geno: %d\n", tid, i, *(&thread_geno[0] + i));
			}
		}
	
	
	// CASE no CV
	if (CV < 0){
		
		printf("a\n");
		
		struct controlscases thread_table[3][3][3];
		
		//printf("b\n");
	
		//replace this initialization?
		for (int i=0; i< 3; i++) {
			for (int j=0; j< 3; j++) {
				for (int k=0; k< 3; k++) {
					thread_table[i][j][k].controls = 0;
					thread_table[i][j][k].cases = 0;
					}}}
		
		
		printf("c\n");
		
		//populate the 3^ORDER-tot-entries table
		int f,s,t,ind;
		for (int n=0; n< NIND; n++) {
			 ind = *(dev_cv_indices + n);
			 f = *(&thread_geno[0] + 0 * NIND + ind); //1st snp geno
			 s = *(&thread_geno[0] + 1 * NIND + ind); //2nd snp geno
			 t = *(&thread_geno[0] + 2 * NIND + ind); //3rd snp geno
			 if (int(*(dev_v_pheno + ind))) //get the pheno
			 	thread_table[f][s][t].cases += 1;
			 else
			 	thread_table[f][s][t].controls += 1;
		}
		__syncthreads();
		printf("d\n");
		//only a print
	
		if (tid == TESTCOMB){
		printf("\n***************\nthread %d\n:", tid);
		for (int i=0; i< 3; i++) {
			for (int j=0; j< 3; j++) {
				for (int k=0; k< 3; k++) {
					printf("thread_table[%d][%d][%d].controls=%d\n",i,j,k,thread_table[i][j][k].controls);
					printf("thread_table[%d][%d][%d].cases=%d\n",i,j,k,thread_table[i][j][k].cases);
					printf("\n");
				
				}
			}
		}
		}
		
		printf("e\n");
	
	
		//moving two a two-dim variable
		int high_cases = 0;
		int high_controls = 0;
		int low_cases = 0;
		int low_controls = 0;
		int high_genos[3*3*3][3]; //content: 001,021,112,..,9xx,..: 3**ORDER strings, each 3 chars
		int c = 0;
		for (int i=0; i< 3; i++) {
			for (int j=0; j< 3; j++) {
				for (int k=0; k< 3; k++) {
					if ((thread_table[i][j][k].cases)/(thread_table[i][j][k].controls + 0.01) >= THR){
						high_cases += thread_table[i][j][k].cases;
						high_controls += thread_table[i][j][k].controls;
						if (tid == TESTCOMB){
							printf("tid %d (comb. <%d, %d, %d>),"
							" geno %d%d%d is HIGH\n",
							tid, *(&thread_combination[0]), *(&thread_combination[0] + 1),
							*(&thread_combination[0] + 2), i, j ,k);
						}
					
						high_genos[c][0] = i;
						high_genos[c][1] = j;
						high_genos[c][2] = k;
						c+=1;
					}
					else{
						//here in LOW also the case 0 controls 0 cases
						low_cases += thread_table[i][j][k].cases;
						low_controls += thread_table[i][j][k].controls;
						if (tid == TESTCOMB){
							printf("tid %d (comb. <%d, %d, %d>),"
							" geno %d%d%d is LOW\n",
							tid, *(&thread_combination[0]), *(&thread_combination[0] + 1),
							*(&thread_combination[0] + 2), i, j ,k);
						}
					
					}
				}
			}
		}
		high_genos[c][0] = 9; //end sequence, since high_genos only reports the high ones
	
		//printf("******************\n");
		float train_error = float(high_controls + low_cases)/float(high_cases + high_controls + low_cases + low_controls);
	
	
		if (tid == TESTCOMB){
		printf("snp comb. <%d, %d, %d> (tid %d) TRAIN error %1.5f = (%d+%d)/(%d+%d+%d+%d)\n", 
				*(&thread_combination[0] + 0), *(&thread_combination[0] + 1), *(&thread_combination[0] + 2), tid,
				train_error, high_controls, low_cases, high_cases, high_controls, low_cases, low_controls);
		}

		//write result to global memory
		*(dev_output + NUMCOMBS * 0 + 2 * tid + 0) = train_error;
	
	
	
	}
	
	
	
	//CV loop
	for (int cv=0; cv<CV; cv++){
		

		if (tid == TESTCOMB){
			printf("\n******************************\nthread %d, iteration of CV %d/%d:\n\n", tid, cv+1, CV);
			printf("train interval:(0;%d) U (%d;%d)\n", int((cv/float(CV))*NIND), int(((cv+1)/float(CV))*NIND), NIND);
			printf("test interval: [%d;%d]\n", int((cv/float(CV))*NIND), int(((cv+1)/float(CV))*NIND));
		}
	
		struct controlscases thread_table[3][3][3];
	
		//replace this initialization?
		for (int i=0; i< 3; i++) {
			for (int j=0; j< 3; j++) {
				for (int k=0; k< 3; k++) {
					thread_table[i][j][k].controls = 0;
					thread_table[i][j][k].cases = 0;
					}}}
	
		//populate the 3^ORDER-tot-entries table
		int f,s,t,ind;
		for (int n=0; n< NIND; n++) { //first NIND_TRAIN of NIND are for train
			 if ((n >= int((cv/float(CV))*NIND)) && (n <= ((cv+1)/float(CV))*NIND )) //reserved for test
			 	continue;
			 ind = *(dev_cv_indices + n);
			 f = *(&thread_geno[0] + 0 * NIND + ind); //1st snp geno
			 s = *(&thread_geno[0] + 1 * NIND + ind); //2nd snp geno
			 t = *(&thread_geno[0] + 2 * NIND + ind); //3rd snp geno
			 if (int(*(dev_v_pheno + ind))) //get the pheno
			 	thread_table[f][s][t].cases += 1;
			 else
			 	thread_table[f][s][t].controls += 1;
		}
	
	
		//only a print
	
		if (tid == TESTCOMB){
		printf("\n***************\nthread %d\n:", tid);
		for (int i=0; i< 3; i++) {
			for (int j=0; j< 3; j++) {
				for (int k=0; k< 3; k++) {
					printf("thread_table[%d][%d][%d].controls=%d\n",i,j,k,thread_table[i][j][k].controls);
					printf("thread_table[%d][%d][%d].cases=%d\n",i,j,k,thread_table[i][j][k].cases);
					printf("\n");
				
				}
			}
		}
		}
	
	
	
		//moving two a two-dim variable
		int high_cases = 0;
		int high_controls = 0;
		int low_cases = 0;
		int low_controls = 0;
		int high_genos[3*3*3][3]; //content: 001,021,112,..,9xx,..: 3**ORDER strings, each 3 chars
		int c = 0;
		for (int i=0; i< 3; i++) {
			for (int j=0; j< 3; j++) {
				for (int k=0; k< 3; k++) {
					if ((thread_table[i][j][k].cases)/(thread_table[i][j][k].controls + 0.01) >= THR){
						high_cases += thread_table[i][j][k].cases;
						high_controls += thread_table[i][j][k].controls;
						if (tid == TESTCOMB){
							printf("tid %d (comb. <%d, %d, %d>),"
							" geno %d%d%d is HIGH\n",
							tid, *(&thread_combination[0]), *(&thread_combination[0] + 1),
							*(&thread_combination[0] + 2), i, j ,k);
						}
					
						high_genos[c][0] = i;
						high_genos[c][1] = j;
						high_genos[c][2] = k;
						c+=1;
					}
					else{
						//here in LOW also the case 0 controls 0 cases
						low_cases += thread_table[i][j][k].cases;
						low_controls += thread_table[i][j][k].controls;
						if (tid == TESTCOMB){
							printf("tid %d (comb. <%d, %d, %d>),"
							" geno %d%d%d is LOW\n",
							tid, *(&thread_combination[0]), *(&thread_combination[0] + 1),
							*(&thread_combination[0] + 2), i, j ,k);
						}
					
					}
				}
			}
		}
		high_genos[c][0] = 9; //end sequence, since high_genos only reports the high ones
	
		//printf("******************\n");
		float train_error = float(high_controls + low_cases)/float(high_cases + high_controls + low_cases + low_controls);
	
	
		if (tid == TESTCOMB){
		printf("snp comb. <%d, %d, %d> (tid %d) TRAIN error %1.5f = (%d+%d)/(%d+%d+%d+%d)\n", 
				*(&thread_combination[0] + 0), *(&thread_combination[0] + 1), *(&thread_combination[0] + 2), tid,
				train_error, high_controls, low_cases, high_cases, high_controls, low_cases, low_controls);
		}

		//write result to global memory
		*(dev_output + NUMCOMBS * cv + 2 * tid + 0) = train_error;
	
		//*****************
		//TESTING
		//*****************
		int high_cases_test = 0;
		int high_controls_test = 0;
		int low_cases_test = 0;
		int low_controls_test = 0;
		int counter = 0;
		for (int n=0; n< NIND; n++) {
			if (tid == -1)
				printf("cv, n: %d, %d....  \n", cv, n);
			 if ((n < int((cv/float(CV))*NIND)) || (n > int(((cv+1)/float(CV))*NIND)) )//reserved for training
			 	continue;
			 if (tid == -1)
				printf(" accepted!  \n");
				counter +=1;
			 ind = *(dev_cv_indices + n);
			 f = *(&thread_geno[0] + 0 * NIND + ind); //1st snp geno
			 s = *(&thread_geno[0] + 1 * NIND + ind); //2nd snp geno
			 t = *(&thread_geno[0] + 2 * NIND + ind); //3rd snp geno
			 int ph = int(*(dev_v_pheno + ind));
			 //check if fst is in high or low
			 int i;
			 for (i=0; i< (3*3*3); i++) {
			 	 if (high_genos[i][0] == 9)
			 	 	break;
				 if  (high_genos[i][0] == f && high_genos[i][1] == s && high_genos[i][2] == t){
				 	if (ph)
				 		high_cases_test += 1;
				 	else
				 		high_controls_test += 1;
				 	break;
				 	
				 }
			}
		 	 //not in high_cases, it's low
		 	 if (high_genos[i][0] == 9){ //not found in high
			 	 if (ph)
				 	low_cases_test += 1;
				 else
				 	low_controls_test += 1;
			 }
			 
			
	
		}
		
		if (tid == TESTCOMB + 1) printf("cv, COUNTER: %d, %d....  \n", cv, counter);
		//printf("******************\n");
		float test_error = float(high_controls_test + low_cases_test)/float(high_cases_test + high_controls_test + low_cases_test + low_controls_test);
	
	
		if (tid == TESTCOMB){
		printf("snp comb. <%d, %d, %d> (tid %d) TEST error %1.5f = (%d+%d)/(%d+%d+%d+%d)\n", 
				*(&thread_combination[0] + 0), *(&thread_combination[0] + 1), *(&thread_combination[0] + 2), tid,
				test_error, high_controls_test, low_cases_test, high_cases_test, high_controls_test, low_cases_test, low_controls_test);
		}
		
		//write result to global memory
		*(dev_output + NUMCOMBS * cv + 2 * tid + 1) = test_error;
	

	}

}



void parseArgs(int argc, char **argv){
  int i=1;
  if(argc <= 1){
  
    printf("\nusage example: \n  ./MDR_main9 -cf \"../combinations7\" -n_combs 3200000 -gf \"../geno7\" -n_inds 6000 -n_snps 20000 -pf \"../pheno7\" -ord 3 -thr 1 -cv 1 -bs 256 -out \"../out7\"  \n\n");
    printf("\tcf      = combinations file, see README\n");
    printf("\tn_combs			= number of combinations\n");
    printf("\tgf      = genotype file, see README\n");
    printf("\tn_inds			= number of individuals\n");
    printf("\tn_snps			= number of SNPs\n");
    printf("\tpf     = phenotype file, see README\n");
    printf("\tord     = order to test, see README (optional)\n");
    printf("\tthr		= threshold for cases/controls ratio (optional)\n");
    printf("\tcv		= k in k-fold CV (optional)\n");
    printf("\tbs     =  # threads per block\n");
    printf("\tout       = output file\n");
    printf("\thelp       = print this help\n");
	printf("\n\n");
    exit(0);
  }

  while(i<argc){
    if(!strcmp(argv[i], "-gf"))
      genoFile = argv[++i];
    else if(!strcmp(argv[i], "-cf"))
      combFile = argv[++i];
    else if(!strcmp(argv[i], "-pf"))
      phenoFile = argv[++i];
    else if(!strcmp(argv[i], "-out"))
      outputFile = argv[++i];
    else if(!strcmp(argv[i], "-n_snps"))
      NSNPS = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-n_inds"))
      NIND = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-n_combs"))
      NUMCOMBS = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-ord"))
      ORDER = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-thr"))
      THR = atof(argv[++i]);
    else if(!strcmp(argv[i], "-cv"))
      CV = atoi(argv[++i]);
    else if(!strcmp(argv[i], "-out"))
      outputFile = argv[++i];
    else if(!strcmp(argv[i], "-bs"))
      //BSx = 12500; //atoi(argv[++i]);
      ++i;
    else if(!strcmp(argv[i], "-help") || !strcmp(argv[i], "-h")){
    printf("\nusage example: \n  ./MDR_main9 -cf \"../combinations7\" -n_combs 3200000 -gf \"../geno7\" -n_inds 6000 -n_snps 20000 -pf \"../pheno7\" -ord 3 -thr 1 -cv 1 -bs 256 -out \"../out7\"  \n\n");
	    printf("cf      = combinations file, see README\n");
	    printf("n_combs			= number of combinations\n");
	    printf("gf      = genotype file, see README\n");
	    printf("n_inds			= number of individuals\n");
	    printf("n_snps			= number of SNPs\n");
	    printf("pf     = phenotype file, see README\n");
	    printf("ord     = order to test, see README (optional)\n");
	    printf("thr		= threshold for cases/controls ratio (optional)\n");
	    printf("cv		= k in k-fold CV (optional)\n");
	    printf("bs     =  number of threads per block\n");
	    printf("out       = output file\n");
	    printf("help       = print this help\n");
	    printf("\n\n");
    }
     
   
    else{
      fprintf(stderr,"%s : argument not valid! \n",argv[i]);
      exit(1);
    }
    i++;
  }

  if( !genoFile || !phenoFile || !combFile || !outputFile){
    fprintf(stderr,"no files specified	.. exiting\n");
    exit(1);
  }
  return;

}


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "ERROR HANDLED: %s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
  	}
  }
  
void readintData(char *dataFile, unsigned int rows, unsigned int cols, int * data){
  FILE *fp;
  int *dp = data;
  int i;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    //exit(1);
  } 
  
  for (i=0; i<rows*cols; ++i){
	  fscanf(fp, "%d", dp);
	  dp++;
  } 
  fclose(fp);
  return;
}


void readCombinations(char *dataFile, int rows, int cols, int * data){
  FILE *fp;
  int *dp = data;
  int i;

  fp = fopen(dataFile,"r");
  if(fp==NULL){
    fprintf(stderr,"error opening file.. exiting\n");
    //exit(1);
  } 
  
  for (i=0; i<rows*cols; ++i){
	  fscanf(fp, "%d", dp);
	  dp++;
  } 
  fclose(fp);
  return;
}
  	
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

void print_cudaGetDeviceProperties(){
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );
	
	if (count == 0) {
        	printf("error in print_cudaGetDeviceProperties: no devices supporting CUDA.\n");
        	return;
    	}
	
	for (int i=0; i< count; i++) {
		HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
		printf( "   --- General Information for device %d ---\n", i );
		printf( "Name:  %s\n", prop.name );
		printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
		printf( "Clock rate:  %d\n", prop.clockRate );
		printf( "Device copy overlap:  " );
		if (prop.deviceOverlap)
		    printf( "Enabled\n" );
		else
		    printf( "Disabled\n");
		printf( "Kernel execution timeout :  " );
		if (prop.kernelExecTimeoutEnabled)
		    printf( "Enabled\n" );
		else
		    printf( "Disabled\n" );

		printf( "   --- Memory Information for device %d ---\n", i );
		printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
		printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
		printf( "Max mem pitch:  %ld\n", prop.memPitch );
		printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

		printf( "   --- MP Information for device %d ---\n", i );
		printf( "Multiprocessor count:  %d\n",
		            prop.multiProcessorCount );
		printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
		printf( "Registers per mp:  %d\n", prop.regsPerBlock );
		printf( "Threads in warp:  %d\n", prop.warpSize );
		printf( "Max threads per block:  %d\n",
		            prop.maxThreadsPerBlock );
		printf( "Max thread dimensions:  (%d, %d, %d)\n",
		            prop.maxThreadsDim[0], prop.maxThreadsDim[1],
		            prop.maxThreadsDim[2] );
		printf( "Max grid dimensions:  (%d, %d, %d)\n",
		            prop.maxGridSize[0], prop.maxGridSize[1],
		            prop.maxGridSize[2] );
		printf( "\n" );
	}
	return;
}

/************************/
//MAIN
/************************/

int main(int argc, char **argv)
{

	//Parsing the input parameters
	parseArgs(argc,argv);
	 
	 
  	//print_cudaGetDeviceProperties(); 
  	int dev_id = 1;
  	HANDLE_ERROR( hipSetDevice( dev_id ) );
  	
  	printf("*****************\n");
	printf("Multifactor Dimensionality Reduction\n");
	printf("*****************\n");	
	
	int* combinations = (int*)malloc(NUMCOMBS * ORDER * sizeof(int));
	int* dev_combinations;
	//int* dev_cv_indices;
  	int* cv_indices = (int*)malloc(NIND * sizeof(int));
  	int* dev_mat_SNP;
	float* dev_output;
	
	if (CV < 0){
		printf("no input cv; will run only one pass, no train-test... \n");
	}

	//generate a permutation of the individuals indices
	for(int i=0;i<NIND;++i){
        	*(cv_indices + i) = i;
    		}
    		
		//permute r with Fisher-Yates shuffling algorithm
	for (int i = NIND; i >= 0; --i){
		//generate a random number [0, n-1]
		int j = rand() % (i+1);

		//swap the last element with element at random index
		int temp = *(cv_indices + i);
		*(cv_indices + i) = *(cv_indices + j);
		*(cv_indices + j) = temp;
	}

	
	//Allocate host memory 
	int* mat_SNP = (int*)malloc(mat_SNP_size); 
	int* v_pheno = (int*)malloc(v_pheno_size);
	float* output = (float*)malloc(output_size);
  	
  	//Read the matrix in host data
	readintData(genoFile, NSNPS, NIND, mat_SNP);
	printf("geno file read..\n");
	readintData(phenoFile, NIND, 1, v_pheno);
	printf("pheno file read..\n");
	
	if (THR < 0){
		int c = 0;
		for (int i = 0; i < NIND; i++){
			if ( *(v_pheno + i) )
				c += 1;
		}
		THR = float(c)/NIND;
		printf("no input threshold; automatically set to %f = %d/%d \n", THR, c, NIND);
	}
	
	
	
	//Read combinations
	readCombinations(combFile, NUMCOMBS, ORDER, combinations);
	printf("combinations file read..\n");

  	
  	//Allocate device memory
	hipMalloc((void**)&dev_mat_SNP, mat_SNP_size);
	//hipMalloc((void**)&dev_v_pheno.values, dev_v_pheno.mem_size);
	
	hipMalloc((void**)&dev_output, output_size);
  	hipMalloc((void**)&dev_combinations, combinations_size  );
  	hipMalloc((void**)&dev_cv_indices, indices_size);
  	
  	// Copy host memory to device
  	//HANDLE_ERROR( hipMemcpy(dev_v_pheno.values, v_pheno.values, dev_v_pheno.mem_size, hipMemcpyHostToDevice));
  	hipMemcpyToSymbol(HIP_SYMBOL( dev_v_pheno),  v_pheno,  v_pheno_size );
  	hipMemcpyToSymbol(HIP_SYMBOL( dev_cv_indices),  cv_indices,  indices_size );
	HANDLE_ERROR( hipMemcpy(dev_mat_SNP, mat_SNP, mat_SNP_size, hipMemcpyHostToDevice));
  	HANDLE_ERROR( hipMemcpy(dev_combinations, combinations, combinations_size, hipMemcpyHostToDevice));
  	//HANDLE_ERROR( hipMemcpy(dev_cv_indices, cv_indices, indices_size, hipMemcpyHostToDevice));
  	fprintf(stderr,"matrices copied  to GPU \n");
  	
  	
  	
  	//hipHostAlloc((void**)&output.values,output.mem_size,hipHostMallocDefault);
  	
  	// kernel call
	dim3 dimBlock(BSx);//,BSy,BSz);
	dim3 dimGrid(GSx);//,GSy,GSz);
	
	printf("\ncalling the kernel with this configuration:\n");
	printf(" interaction order: %d\n NSNPS: %d\n NIND: %d\n # cross validations: %d\n THRESHOLD: %f\n GRID SIZE: %d\n BLOCK SIZE: %d\n",ORDER, NSNPS, NIND, CV, THR, GSx, BSx);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime;
	hipEventRecord(start, 0);

	MDR<<< dimGrid, dimBlock >>>(dev_mat_SNP, dev_output, dev_combinations, NSNPS, NIND, THR, NUMCOMBS, ORDER, CV);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("kernel computation terminated, Time required (ms): %4.5f\n", elapsedTime);
	
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );

	hipMemcpy(output, dev_output, output_size, hipMemcpyDeviceToHost);
	fprintf(stderr,"Output copied to host \n");
	
	FILE *fpout;
	fpout = fopen(outputFile, "w");
	
	if (CV < 0){
		fprintf(fpout,"---------- error ----------\n");
		for (int j = 0; j < NUMCOMBS; j++){
			fprintf(fpout,"<snp%d,snp%d,snp%d> %f\n", *(combinations + j),*(combinations + j + 1), 
			*(combinations + j + 2), *(output + 0 * NUMCOMBS + 2 * j));
		}
	}
	
  	for (int cv = 0; cv < CV; cv++){
  		fprintf(fpout,"---------- CV %d/%d train_error test_error ----------\n", cv+1, CV);
		for (int j = 0; j < NUMCOMBS; j++){
			fprintf(fpout,"<snp%d,snp%d,snp%d> %f %f\n", *(combinations + j),*(combinations + j + 1), 
			*(combinations + j + 2), *(output + cv * NUMCOMBS + 2 * j), *(output + cv * NUMCOMBS + 2 * j + 1));
		
		}
	}
  	
  	
  	
  	
  	
  	//free
  	hipFree(dev_mat_SNP);
	//hipFree(dev_v_pheno.values);
	hipFree(dev_output);
	hipFree(dev_combinations);
	//hipFree(dev_cv_indices);

	free(mat_SNP);
	free(v_pheno);
	free(output);
	free(combinations);
	free(cv_indices);

  	
  	
  	
  	
  	
  	
  	
 	return 0;
}
